using namespace std;

#include <stdio.h>
#include "fstream"
#include <stdlib.h>
#include <stddef.h>
#include <math.h>
#include <vector>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void SAXS(float* X, float* q, float* F, float* Iq)
{
	int N = 2140;
	int k = blockIdx.x*blockDim.x + threadIdx.x;

	float eucdist = 0;
	float sincin = 0;

	if (k < 60){
		for (int i = 0; i<N; i++){
			for(int j = 0; j<N; j++){

				eucdist = 0;
				
				eucdist = eucdist + ((X[i]-X[j])*(X[i]-X[j]));
				eucdist = eucdist +((X[i+N]-X[j+N])*(X[i+N]-X[j+N]));
				eucdist = eucdist + ((X[i+(2*N)]-X[j+(2*N)])*(X[i+(2*N)]-X[j+(2*N)]));

				eucdist = sqrt(eucdist);

				sincin = q[k]*eucdist;
				
				if(sincin == 0)
					Iq[k] = Iq[k] + (F[i+(N*k)] * F[j+(N*k)] * 1);	
				else
					Iq[k] = Iq[k] + (F[i+(N*k)] * F[j+(N*k)] * sin(sincin)/sincin);	

			}

		}
		__syncthreads();	
	}
}


int main(int argc, char **argv)
{
  	hipEvent_t start;
  	hipEvent_t stop;
  	float time;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);


	int N = 2140;
	int Q = 60;

	float *X, *q, *F, *Iq;
	float *X_d, *q_d, *F_d, *Iq_d;	

	int SIZE_X = N*3*sizeof(float);
	int SIZE_q = Q*sizeof(float);
	int SIZE_F = N*Q*sizeof(float);
	int SIZE_Iq = Q*sizeof(float);

	int DimBlock = 512;	
	int DimGrid = (int)ceil((float)SIZE_Iq/DimBlock);

	X = (float*)malloc(SIZE_X);
	q = (float*)malloc(SIZE_q);
	F = (float*)malloc(SIZE_F);
	Iq = (float*)malloc(SIZE_Iq);

	hipMalloc(&X_d, SIZE_X);
	hipMalloc(&q_d, SIZE_q);
	hipMalloc(&F_d, SIZE_F);
	hipMalloc(&Iq_d, SIZE_Iq);

	// import data files
	int i,j;

	ifstream input("X.txt");
	for(i = 0; i<N; i++){
		for(j = 0; j<3; j++){
			input>>X[i+(j*N)];
		}
	}
	input.close();

	ifstream input2("q.txt");
	for(i = 0; i<Q; i++){
		input2>>q[i];	
	}
	input2.close();

	ifstream input3("F.txt");
	for(i = 0; i<N; i++){
		for(j = 0; j<Q; j++){
			input3>>F[i+(N*j)];
		}
	}
	input3.close();


	hipMemcpy(X_d, X, SIZE_X, hipMemcpyHostToDevice);
	hipMemcpy(q_d, q, SIZE_q, hipMemcpyHostToDevice);
	hipMemcpy(F_d, F, SIZE_F, hipMemcpyHostToDevice);
	hipMemcpy(Iq_d, Iq, SIZE_Iq, hipMemcpyHostToDevice);

  	hipEventRecord(start,0);

	SAXS<<<DimGrid,DimBlock>>>(X_d, q_d, F_d, Iq_d);

  	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
  	
	hipMemcpy(Iq, Iq_d, SIZE_Iq, hipMemcpyDeviceToHost);

	hipFree(X_d);
	hipFree(q_d);
	hipFree(F_d);
	hipFree(Iq_d);

	hipEventElapsedTime(&time,start,stop);
	printf("Time for the kernel: %f ms\n", time);

}

